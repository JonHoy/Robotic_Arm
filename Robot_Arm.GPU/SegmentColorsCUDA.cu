
#include <hip/hip_runtime.h>
//#include <cuda.h>
//#include <cuda_runtime.h>
//#include <math.h>
//#include "NativeDeclarations.h"
//#include <memory>
//
//#define PI 3.14159f
//#define BlockSize 16
//#define IterationLimit 10
//#define Planes 3
//
//template<typename T, int BLOCK_SIZE>
//__global__ void MeanFilterKernel(int Rows, int Cols, T* Image, int NumIterations)
//{
//    // Thread index
//    int tx = threadIdx.x;
//    int ty = threadIdx.y;
//
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//
//	// put padding in for the edge (do all calcs in float and then convert back to native format)
//	__shared__ float OriginalImageTile[BLOCK_SIZE + 2][BLOCK_SIZE + 2][Planes];
//	
//	__shared__ float NewImageTile[BLOCK_SIZE][BLOCK_SIZE][Planes];
//
//	for (int iPlane = 0; iPlane < Planes; iPlane++)
//	{
//		int idx = (x + y * blockDim.x * gridDim.x) * Planes + iPlane;
//		for (int i = 0; i < NumIterations; i++)
//		{	
//			float val = (float) Image[idx];
//			if (idx < Rows * Cols * Planes)
//			{
//				if (tx == 0)
//					OriginalImageTile[0][ty + 1][iPlane] = val;	
//				if (ty == 0)
//					OriginalImageTile[tx + 1][0][iPlane] = val;
//				if (tx == blockDim.x)
//					OriginalImageTile[tx + 2][ty + 1][iPlane] = val;
//				if (ty == blockDim.y)
//					OriginalImageTile[tx + 1][ty + 2][iPlane] = val;
//
//				OriginalImageTile[tx + 1][ty + 1][iPlane] = val;
//			}
//			__syncthreads();
//			NewImageTile[tx][ty][iPlane] = (
//			OriginalImageTile[tx + 1][ty + 1][iPlane] +
//			OriginalImageTile[tx][ty + 1][iPlane] +
//			OriginalImageTile[tx + 2][ty + 1][iPlane] +
//			OriginalImageTile[tx + 1][ty][iPlane] +
//			OriginalImageTile[tx + 1][ty + 2][iPlane])/5;
//			__syncthreads();
//		}
//		Image[idx] = (T)NewImageTile[tx + 1][ty + 1][iPlane];
//	}
//			
//}
//
//// helper routine to convert colorspace from rgb to hsi
//__host__ __device__ void RGB2HSI(float R, float G, float B, float* H, float* S, float* I) {
//	*I = (R + G + B)/3;
//	float MinColor;
//	
//	if (R < G)
//		MinColor = R;
//	else
//		MinColor = G;
//	if (MinColor > B)
//		MinColor = B;
//	
//	*S = 1 - MinColor / *I;
//	
//	float Theta;
//	if (R == 0 && G == 0 && B == 0)
//		Theta = 0;
//	else
//		Theta = acosf(((R - G) + (R - B))/ 2 / sqrtf((R - G)*(R - G) + (R - B)*(G - B))); 
//	if (B <= G)
//		*H = Theta/(2 * PI);
//	else
//		*H = ((2 * PI) - Theta)/(2*PI);
//}
//
//template <typename T>
//__global__ void SegmentKernel(T* Image, int Rows, int Cols, float* Colors, int NumColors, int* SelectedColors) {
//	float R; // red value (0 to 1)
//	float G; // blue value (0 to 1)
//	float B; // green value (0 to 1)
//	int x = threadIdx.x + blockIdx.x * blockDim.x;
//	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int idx = (x + y * blockDim.x * gridDim.x);
//	R = ((float) Image[idx * Planes]) / 255.0f;
//	G = ((float) Image[idx * Planes + 1]) / 255.0f;
//	B = ((float) Image[idx * Planes + 2]) / 255.0f;
//	float H;
//	float S;
//	float I;
//	RGB2HSI(R, G, B, &H, &S, &I);
//	float MinDistance = 3;
//	float Distance;
//	int ClosestColor = 0;
//	for (int iColor = 0; iColor < NumColors; iColor++)
//	{
//		float DistH = H - Colors[iColor * Planes];
//		float DistS = S - Colors[iColor * Planes + 1];
//		float DistI = H - Colors[iColor * Planes + 2];
//		Distance = DistH * DistH + DistI * DistI + DistS * DistS;
//		if (Distance < MinDistance) {
//			MinDistance = Distance;
//			ClosestColor = iColor;
//		}		
//	}
//	SelectedColors[idx] = ClosestColor;
//}
//// RGB values are 0-1 and HSI outputs 0-1
//
//namespace native_library {
//	namespace details{
//		template<typename T>
//		void SegmentColorsCUDA(T* Image, int Rows, int Columns, T* Colors, int NumColors, int* SelectedColors)
//		{
//			cudaError Status;
//			int PixelCount = Rows * Columns;
//			//T* dev_Image;
//			//float* dev_Colors;
//			int* dev_SelectedColors;
//			float* Colors_float;
//			Colors_float = (float*) malloc(NumColors * Planes * sizeof(float));
//			for (int iColor = 0; iColor < NumColors; iColor++)
//			{
//				float R = ((float) Colors[iColor * Planes]) / 255.0f;
//				float G = ((float) Colors[iColor * Planes + 1]) / 255.0f;
//				float B = ((float) Colors[iColor * Planes + 2]) / 255.0f;
//				RGB2HSI(R, G, B, &Colors_float[iColor * Planes] , &Colors_float[iColor * Planes + 1], &Colors_float[iColor * Planes + 2]);
//				printf("H: %f",Colors_float[iColor * Planes]);
//				printf(" |S: %f",Colors_float[iColor * Planes + 1]);
//				printf(" |I: %f\n",Colors_float[iColor * Planes + 2]);
//			}
//			//Status = cudaMalloc((void **) &dev_Image, PixelCount * Planes * sizeof(T));
//			Status = cudaMalloc((void **) &dev_SelectedColors,  PixelCount * sizeof(int));
//			//Status = cudaMalloc((void **) &dev_Colors, NumColors * Planes * sizeof(float));
//			//Status = cudaMemcpy(dev_Image, Image, PixelCount * Planes * sizeof(T), cudaMemcpyHostToDevice);
//			//Status = cudaMemcpy(dev_Colors, Colors_float, NumColors * Planes * sizeof(float), cudaMemcpyHostToDevice);
//			dim3 TileSize(BlockSize, BlockSize);
//			dim3 GridSize(Rows/BlockSize, Columns/BlockSize);
////			 Filter image data on gpu side before processing
//			//MeanFilterKernel<T, BlockSize><<<TileSize, GridSize>>>(Rows, Columns, dev_Image, IterationLimit);
//			//SegmentKernel<T><<<TileSize, GridSize>>>(dev_Image, Rows, Columns, dev_Colors, NumColors, dev_SelectedColors);
//			//Status = cudaMemcpy(SelectedColors, dev_SelectedColors, PixelCount * sizeof(int), cudaMemcpyDeviceToHost);
//			free(Colors_float);
//			cudaFree(dev_SelectedColors);
//			//cudaFree(dev_Image);
////			cudaFree(dev_Colors);
//		}
//	}
//
//	void SegmentColorsCUDA(float* Image, int Rows, int Columns, float* Colors, int NumColors, int* SelectedColors) {
//		details::SegmentColorsCUDA<float>(Image, Rows, Columns, Colors, NumColors, SelectedColors);
//	}
//
//	void SegmentColorsCUDA(int* Image, int Rows, int Columns, int* Colors, int NumColors, int* SelectedColors) {
//		details::SegmentColorsCUDA<int>(Image, Rows, Columns, Colors, NumColors, SelectedColors);
//	}
//
//	void SegmentColorsCUDA(unsigned char* Image, int Rows, int Columns, unsigned char* Colors, int NumColors, int* SelectedColors) {
//		details::SegmentColorsCUDA<unsigned char>(Image, Rows, Columns, Colors, NumColors, SelectedColors);
//	}
//
//}
