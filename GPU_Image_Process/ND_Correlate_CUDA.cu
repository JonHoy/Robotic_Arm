#include "hip/hip_runtime.h"
#include "ND_Correlate_CUDA.h"


__global__ void ndCorrelateKernel(
	float* Y,
	float* X,
	float* Range,
	int* Bins,
	int* Cumprod,
	int Dims,
	int numPoints,
	float* Sum,
	float* Max,
	float* Min,
	int* Count,
	int* Index
	)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int idx = 0;
	while (i < numPoints)
	{
		for (int iDim = 0; iDim < Dims; iDim++)
		{
			float Nbins = (float) Bins[iDim];
			float RawIndex = (X[i * Dims + iDim] - Range[iDim * Dims]) / (Range[iDim * Dims + 1] - Range[iDim * Dims]) * Nbins;
			//int LocalIndex = (int) floorf(RawIndex);
			int LocalIndex = (int) RawIndex;
			//			 if index is out of bounds clip the value
				if (LocalIndex < 0)
					LocalIndex = 0;
				else if (LocalIndex >= Bins[iDim])
					LocalIndex = Bins[iDim] - 1;
				idx = idx + Cumprod[iDim] * LocalIndex;
		}
		Index[i] = idx;
		atomicAdd(&Count[idx],1); // add to bin count
		atomicAdd(&Sum[idx],Y[i]); // keep a running sum
		if (Max[idx] < Y[i] || i == 0)
			atomicExch(&Max[idx], Y[i]);
		if (Min[idx] > Y[i] || i == 0)
			atomicExch(&Min[idx], Y[i]); 
	}
}

// Kernel to calculate the standard deviation of the dataset

__global__ void ndCorrelateKernel_STD(
	float* Y,
	int numPoints,
	float* Sum,
	float* Std,
	int* Count,
	int* Index
	)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < numPoints)
	{
		int idx = Index[i];
		float var = (Sum[idx]/Count[idx] - Y[i]) * (Sum[idx]/Count[idx] - Y[i]); // keep a running sum of the variance
		atomicAdd(&Std[idx],var); // add up the variance
		i += stride;
	}
	__syncthreads(); // now synchronize all threads so that we can now compute the standard deviation
}

extern "C" __declspec ( dllexport ) void _stdcall ND_Correlate_CUDA(
	const float* Y, 
	const float* X, 
	const int Y_length,
	const int Dims, 
	const float* Range, 
	const int *Bins,
	float* Max,
	float* Min,
	float* Std,
	float* Avg,
	int* Count)

{
	const int threadsPerBlock = 256;
	const int blocksPerGrid = max(32, Y_length/threadsPerBlock);

	int *Cumprod;
	Cumprod = (int*)malloc(Dims*sizeof(int));
	Cumprod[0] = 1;
	for (int iDim = 1; iDim < Dims; iDim++)
	{
		Cumprod[iDim] = Bins[iDim] * Cumprod[iDim - 1];
	}


	int *dev_Count, *dev_Cumprod, *dev_Bins, *dev_Index;
	float *dev_Max, *dev_Min, *dev_Var, *dev_Sum, *dev_Range;
	float *dev_X, *dev_Y;

	int NumBins = 1;
	for (int iDim = 0; iDim < Dims; iDim++)
	{
		NumBins = NumBins * Bins[iDim]; // count the total number of Bins needed for the Correlation operations
	}

	// allocate memory on the GPU
	hipMalloc((void**)&dev_Bins, Dims * sizeof(int));
	hipMalloc((void**)&dev_Range, Dims * 2 * sizeof(float));
	hipMalloc((void**)&dev_Cumprod, Dims * sizeof(int));
	hipMalloc((void**)&dev_Count, NumBins * sizeof(int));
	hipMalloc((void**)&dev_Max, NumBins * sizeof(float));
	hipMalloc((void**)&dev_Min, NumBins * sizeof(float));
	hipMalloc((void**)&dev_Sum, NumBins * sizeof(float));
	hipMalloc((void**)&dev_Var, NumBins * sizeof(float));
	hipMalloc((void**)&dev_X, Y_length * Dims * sizeof(float));
	hipMalloc((void**)&dev_Y, Y_length * sizeof(float));
	hipMalloc((void**)&dev_Index, Y_length*sizeof(int));

	// copy memory from the cpu over to the gpu
	hipMemcpy(dev_Bins, Bins, Dims * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Cumprod, Cumprod, Dims * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Count, Count, NumBins * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_X, X, Dims * Y_length* sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Y, Y, Y_length * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Max, Max, NumBins * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Min, Min, NumBins * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Var, Std, NumBins * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Sum, Avg, NumBins * sizeof(float), hipMemcpyHostToDevice);
	
	ndCorrelateKernel<<<threadsPerBlock, blocksPerGrid>>>(dev_Y, dev_X, dev_Range, dev_Bins, dev_Cumprod, Dims, Y_length, dev_Sum, dev_Max, dev_Min, dev_Count, dev_Index);
	// calculate the variance from the Avg 
	ndCorrelateKernel_STD<<<threadsPerBlock, blocksPerGrid>>>(dev_Y, Y_length, dev_Sum, dev_Var, dev_Count, dev_Index);
	
	// copy memory from the gpu back over to the cpu
	hipMemcpy(Max, dev_Max, NumBins * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Min, dev_Min, NumBins * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Avg, dev_Sum, NumBins * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Std, dev_Var, NumBins * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(Count, dev_Count, NumBins * sizeof(float), hipMemcpyDeviceToHost);

	// transform sum of variance and sum to avg and std
	for (int i = 0; i < NumBins; i++)
	{
		if (Count[i] > 0)
		{
			Std[i] = Std[i]/((float) Count[i]);
			Avg[i] = Avg[i]/((float) Count[i]);
		}
	}



	// free memory on the gpu
	hipFree(dev_Range);
	hipFree(dev_Index);
	hipFree(dev_Count);
	hipFree(dev_Bins);
	hipFree(dev_Cumprod);
	hipFree(dev_Sum);
	hipFree(dev_Max);
	hipFree(dev_Min);
	hipFree(dev_Var);
	hipFree(dev_X);
	hipFree(dev_Y);

	free(Cumprod);
}